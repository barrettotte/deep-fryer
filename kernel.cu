#include "hip/hip_runtime.h"
#include "kernel.cuh"

#define TX 32
#define TY 32
#define RAD 1

int divUp(int a, int b) {
    return (a + b - 1) / b;
}

__device__ unsigned char clip(int n) {
    return n > 255 ? 255 : (n < 0 ? 0 : n);
}

__device__ int idxClip(int idx, int idxMax) {
    return idx > (idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx);
}

__device__ int flatten(int col, int row, int width, int height) {
    return idxClip(col, width) + idxClip(row, height) * width;
}

__global__ void sharpenKernel(uchar4* d_out, const uchar4* d_in, const float* d_filter, int w, int h) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((x >= w) || (y >= h)) {
        return;
    }
    const int i = flatten(x, y, w, h);
    const int filterSize = 2 * RAD + 1;
    float rgb[3] = {0.0f, 0.0f, 0.0f};

    // apply filter to each pixel
    for (int yd = -RAD; yd <= RAD; yd++) {
        for (int xd = -RAD; xd <= RAD; xd++) {
            int imgIdx = flatten(x + xd, y + yd, w, h);
            int filterIdx = flatten(RAD + xd, RAD + yd, filterSize, filterSize);
            uchar4 color = d_in[imgIdx];
            float weight = d_filter[filterIdx];

            rgb[0] += weight * color.x;
            rgb[1] += weight * color.y;
            rgb[2] += weight * color.z;
        }
    }
    d_out[i].x = clip(rgb[0]);
    d_out[i].y = clip(rgb[1]);
    d_out[i].z = clip(rgb[2]);
}

void brighten(uchar4* arr, int w, int h) {
    // TODO:
}

void contrast(uchar4* arr, int w, int h) {
    // TODO:
}

void sharpen(uchar4 *arr, int w, int h) {
    const float filter[9] = {
        -1.0f, -1.0f, -1.0f,
        -1.0f, 9.0f, -1.0f,
        -1.0f, -1.0f, -1.0f
    };
    const int filterSize = 2 * RAD + 1;

    float* d_filter = 0;
    uchar4* d_in = 0;
    uchar4* d_out = 0;

    hipMalloc(&d_filter, filterSize * filterSize * sizeof(float));
    hipMalloc(&d_in, w * h * sizeof(uchar4));
    hipMalloc(&d_out, w * h * sizeof(uchar4));

    hipMemcpy(d_filter, filter, filterSize * filterSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_in, arr, w * h * sizeof(uchar4), hipMemcpyHostToDevice);

    const dim3 blockSize(TX, TY);
    const dim3 gridSize(divUp(w, blockSize.x), divUp(h, blockSize.y));

    sharpenKernel<<<gridSize, blockSize>>>(d_out, d_in, d_filter, w, h);
    hipMemcpy(arr, d_out, w * h* sizeof(uchar4), hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_in);
    hipFree(d_filter);
}

void saturate(uchar4* arr, int w, int h) {
    // TODO:
}

void hueShift(uchar4* arr, int w, int h) {
    // TODO:
}
